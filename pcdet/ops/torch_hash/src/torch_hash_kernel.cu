#include "hip/hip_runtime.h"
#include <stdio.h>
#include "stdint.h"
#include "torch_hash.h"
#include <torch/serialize/tensor.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define CHECK_CUDA(x) do { \
    if (!x.type().is_cuda()) { \
          fprintf(stderr, "%s must be CUDA tensor at %s:%d\n", #x, __FILE__, __LINE__); \
          exit(-1); \
        } \
} while (0)
#define CHECK_CONTIGUOUS(x) do { \
    if (!x.is_contiguous()) { \
          fprintf(stderr, "%s must be contiguous tensor at %s:%d\n", #x, __FILE__, __LINE__); \
          exit(-1); \
        } \
} while (0)
#define CHECK_INPUT(x) CHECK_CUDA(x);CHECK_CONTIGUOUS(x)

//__device__ Key EMPTY = 0xffffffffffffffff;
__device__ Key EMPTY = -1;
__device__ Key rp0 = 999269;
__device__ Key rp1 = 999437;
__device__ Key rp2 = 1999377;

__device__ index_t map2key(const Key* keys, const Key* dims, int num_dim) {
  index_t ans = 0;
  for (int i = 0; i < num_dim; i++) {
    Key key = keys[i];
    //printf("key=%d\n", key);
    if (key >= dims[i]) {
      key = dims[i];
    }
    if (key < 0) {
      key = 0;
    }
    ans = ans * dims[i] + key;
    //printf("ans=%d\n", ans);
  }
  //printf("ans=%d\n", ans);
  return ans;
}

__device__ index_t hashkey(const Key key, index_t ht_size) {
  return ((key % ht_size) * rp0 + rp1) % ht_size;
}

// Insert (key, value) pairs into hash table
__global__ void hash_insert_gpu_kernel(
                  Key* ht_keys,
                  Float* ht_values,
                  Key* reverse_indices,
                  index_t ht_size,
                  const Key* dims,
                  int num_dim,
                  const Key* insert_keys, // keys in [N, D]
                  const Float* insert_values,
                  uint32 num_inserts
                  ) {
  unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadid < num_inserts) {
    const Key* insert_key_ptr = &insert_keys[threadid*num_dim];
    Key insert_key = map2key(insert_key_ptr, dims, num_dim);
    index_t hash_idx = hashkey(insert_key, ht_size);
    
    const Float* insert_value = &insert_values[threadid*num_dim];
    Key prev = atomicCAS((unsigned long long int*)(&ht_keys[hash_idx]),
                         (unsigned long long int)EMPTY,
                         (unsigned long long int)insert_key);
    while (prev != EMPTY) {
      hash_idx = (hash_idx + 1) % ht_size;
      prev = atomicCAS((unsigned long long int*)(&ht_keys[hash_idx]),
                       (unsigned long long int)EMPTY,
                       (unsigned long long int)insert_key);
    }
    if (prev == EMPTY) {
      // hit
      ht_keys[hash_idx] = insert_key;
      Float* ht_value = &ht_values[hash_idx*num_dim];
      for (int i = 0; i < num_dim; i++) {
        ht_value[i] = insert_value[i];
      }
      reverse_indices[hash_idx] = threadid;
    }
  }
}

// for each query points (query_keys, query_values),
//   find corresponding point in hash table (ht_keys, ht_values)
// 
__global__ void correspondence_kernel(
                  Key* ht_keys, // hash table keys, values
                  Float* ht_values,
                  Key* reverse_indices, // indices to original hashed array
                  index_t ht_size, // hashtable size
                  const Key* dims, // maximum size of each dimension
                  int num_dim, // number of dimensions
                  Key* query_keys, // query keys in shape [N, D]
                  const Float* query_values, // query values
                  uint32 num_queries, //
                  const int* qmin, const int* qmax, // query range in each dimension
                  Key* corres_indices // correspondence results
                  ) {
  unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadid < num_queries) {
    Key* query_key_ptr = &query_keys[threadid*num_dim];
    
    // number of points to query
    int num_combination = 1;
    for (int i = 0; i < num_dim; i++) {
      num_combination *= (qmax[i] - qmin[i] + 1);
    }
    Float min_dist = 1e10;
    corres_indices[threadid] = -1;

    // enumerate all directions
    Float dist, di;
    for (int c = 0; c < num_combination; c++) {
      int temp = c;
      for (int i = 0; i < num_dim; i++) {
        query_key_ptr[i] += temp % (qmax[i] - qmin[i] + 1) + qmin[i];
        temp /= (qmax[i] - qmin[i] + 1);
      }
      Key query_key = map2key(query_key_ptr, dims, num_dim);
      index_t hash_idx = hashkey(query_key, ht_size);
      const Float* query_value = &query_values[threadid*num_dim];
      while (ht_keys[hash_idx] != -1) {
        if (ht_keys[hash_idx] == query_key) {
          const Float* ht_value = &ht_values[hash_idx*num_dim];
          // calculate distance
          dist = 0.0;
          for (int i = 0; i < num_dim; i++) {
            di = ht_value[i] - query_value[i];
            dist = dist + di*di;
          }
          if (dist < min_dist) {
            min_dist = dist;
            corres_indices[threadid] = reverse_indices[hash_idx];
          }
        }
        hash_idx = (hash_idx + 1) % ht_size;
      }
      temp = c;
      for (int i = 0; i < num_dim; i++) {
        query_key_ptr[i] -= temp % (qmax[i] - qmin[i] + 1) + qmin[i];
        temp /= (qmax[i] - qmin[i] + 1);
      }
    }
  }
}

// compute points in with radius `radius` of any query points
//   mark them with `visited`
// 
__global__ void points_in_radius_kernel(
                  Key* ht_keys, // hash table keys, values
                  Float* ht_values,
                  Key* reverse_indices, // indices to original hashed array
                  index_t ht_size, // hashtable size
                  const Key* dims, // maximum size of each dimension
                  int num_dim, // number of dimensions
                  Key* query_keys, // query keys in shape [N, D]
                  const Float* query_values, // query values
                  uint32 num_queries, //
                  const int* qmin, const int* qmax, // query range in each dimension
                  const Float radius,
                  Key* visited // correspondence results
                  ) {
  unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadid < num_queries) {
    Key* query_key_ptr = &query_keys[threadid*num_dim];
    
    // number of points to query
    int num_combination = 1;
    for (int i = 0; i < num_dim; i++) {
      num_combination *= (qmax[i] - qmin[i] + 1);
    }

    // enumerate all directions
    Float dist, di;
    Float radius2 = radius*radius;
    for (int c = 0; c < num_combination; c++) {
      int temp = c;
      for (int i = 0; i < num_dim; i++) {
        query_key_ptr[i] += temp % (qmax[i] - qmin[i] + 1) + qmin[i];
        temp /= (qmax[i] - qmin[i] + 1);
      }
      Key query_key = map2key(query_key_ptr, dims, num_dim);
      index_t hash_idx = hashkey(query_key, ht_size);
      const Float* query_value = &query_values[threadid*num_dim];
      while (ht_keys[hash_idx] != -1) {
        if (ht_keys[hash_idx] == query_key) {
          const Float* ht_value = &ht_values[hash_idx*num_dim];
          // calculate distance
          dist = 0.0;
          for (int i = 0; i < num_dim; i++) {
            di = ht_value[i] - query_value[i];
            dist = dist + di*di;
          }

          if (dist < radius2) {
            int reverse_idx = reverse_indices[hash_idx];
            auto prev = atomicCAS((unsigned long long int*)(&visited[reverse_idx]),
                                  (unsigned long long int)0,
                                  (unsigned long long int)1);
          }
        }
        hash_idx = (hash_idx + 1) % ht_size;
      }
      temp = c;
      for (int i = 0; i < num_dim; i++) {
        query_key_ptr[i] -= temp % (qmax[i] - qmin[i] + 1) + qmin[i];
        temp /= (qmax[i] - qmin[i] + 1);
      }
    }
  }
}

__global__ void count_radius_graph_degree_kernel(
                  Key* ht_keys, // hash table keys, values
                  Float* ht_values, // [N, D]
                  Key* reverse_indices, // indices to original hashed array [N]
                  index_t ht_size, // hashtable size
                  const Key* dims, // maximum size of each dimension [D]
                  int num_dim, // number of dimensions D
                  Key* query_keys, // query keys in shape [M, D]
                  const Float* query_values, // query values [M, D]
                  uint32 num_queries, // M
                  const int* qmin, const int* qmax, // query range in each dimension
                  int* degree, // max number of neighbors per query MNN
                  const int max_num_neighbors, // -1 indicate infinity
                  const Float* radius_
                  ) {
  unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadid < num_queries) {
    Key* query_key_ptr = &query_keys[threadid*num_dim];

    int &num_neighbors = degree[threadid];
    num_neighbors = 0;
    // number of points to query
    int num_combination = 1;
    for (int i = 0; i < num_dim; i++) {
      num_combination *= (qmax[i] - qmin[i] + 1);
    }

    // enumerate all directions
    const Float &radius = radius_[threadid];
    Float radius2 = radius*radius;
    for (int c = 0; c < num_combination; c++) {
      int temp = c;
      for (int i = 0; i < num_dim; i++) {
        query_key_ptr[i] += temp % (qmax[i] - qmin[i] + 1) + qmin[i];
        temp /= (qmax[i] - qmin[i] + 1);
      }
      Key query_key = map2key(query_key_ptr, dims, num_dim);
      index_t hash_idx = hashkey(query_key, ht_size);
      const Float* query_value = &query_values[threadid*num_dim];
      while (ht_keys[hash_idx] != -1) {
        if (ht_keys[hash_idx] == query_key) {
          const Float* ht_value = &ht_values[hash_idx*num_dim];
          // calculate distance
          Float dist2 = 0.0;
          for (int i = 0; i < num_dim; i++) {
            Float di = ht_value[i] - query_value[i];
            dist2 = dist2 + di*di;
          }
          if ((dist2 <= radius2) && 
              ((max_num_neighbors == -1) || (num_neighbors < max_num_neighbors))
              ) {
            num_neighbors++;
          }
        }
        hash_idx = (hash_idx + 1) % ht_size;
      }
      temp = c;
      for (int i = 0; i < num_dim; i++) {
        query_key_ptr[i] -= temp % (qmax[i] - qmin[i] + 1) + qmin[i];
        temp /= (qmax[i] - qmin[i] + 1);
      }
    }
  }
  __syncthreads();
}

__global__ void radius_graph_kernel(
                  Key* ht_keys, // hash table keys, values
                  Float* ht_values, // [N, D]
                  Key* reverse_indices, // indices to original hashed array [N]
                  index_t ht_size, // hashtable size
                  const Key* dims, // maximum size of each dimension [D]
                  int num_dim, // number of dimensions D
                  Key* query_keys, // query keys in shape [M]
                  const Float* query_values, // query values [M, D]
                  uint32 num_queries, // M
                  const int* qmin, const int* qmax, // query range in each dimension
                  const int* max_num_neighbors, // max number of neighbors per query
                  const int* offset, // offset of each query in edge array
                  Key* edges, // the edge array [E, 2]
                  const Float* radius_,
                  Float* dists,
                  int max_degree,
                  bool sort_by_dist
                  ) {
  unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadid < num_queries) {
    Key* query_key_ptr = &query_keys[threadid*num_dim];
    
    Key* edges_ptr = &edges[offset[threadid]*2];
    int num_neighbors = 0;
    // number of points to query
    int num_combination = 1;
    for (int i = 0; i < num_dim; i++) {
      num_combination *= (qmax[i] - qmin[i] + 1);
    }

    const int max_num_neighbor = max_num_neighbors[threadid];
    // enumerate all directions
    const Float &radius = radius_[threadid];
    Float radius2 = radius*radius;
    Float* dists_ptr = &dists[offset[threadid]];
    //if (threadid <= 10) {
    //  printf("%d: max_num_neighbor=%d, max_degree=%d, threadidx.x=%u\n", threadid, max_num_neighbor, max_degree, threadIdx.x);
    //}
    for (int c = 0; c < num_combination; c++) {
      int temp = c;
      for (int i = 0; i < num_dim; i++) {
        query_key_ptr[i] += temp % (qmax[i] - qmin[i] + 1) + qmin[i];
        temp /= (qmax[i] - qmin[i] + 1);
        //if (threadid == 0) {
        //  printf("%d: dims(%d)=%d\n", threadid, i, dims[i]);
        //}
        //if (threadid <= 10) {
        //  printf("%d: query_key_ptr(%d)=%d\n", threadid, i, query_key_ptr[i]);
        //}
      }

      //Key query_key = 0;
      //for (int i = 0; i < num_dim; i++) {
      //  Key key = query_key_ptr[i];
      //  if (key >= dims[i]) {
      //    key = dims[i];
      //  }
      //  if (key < 0) {
      //    key = 0;
      //  }
      //  query_key = query_key * dims[i] + key;
      //}
      //printf("%d: ans= %" PRId64 "\n", threadid, query_key);

      Key query_key = map2key(query_key_ptr, dims, num_dim);
      //printf("%d: num_dim=%d\n", threadid, num_dim);
      //printf("%d: querying %" PRId64 "\n", threadid, query_key);
      index_t hash_idx = hashkey(query_key, ht_size);
      const Float* query_value = &query_values[threadid*num_dim];
      while (ht_keys[hash_idx] != -1) {
        if (ht_keys[hash_idx] == query_key) {
          const Float* ht_value = &ht_values[hash_idx*num_dim];
          // calculate distance
          Float dist2 = 0.0;
          for (int i = 0; i < num_dim; i++) {
            Float di = ht_value[i] - query_value[i];
            dist2 = dist2 + di*di;
          }
          //printf("%d: reverse index=%" PRId64 "\n", threadid, reverse_indices[hash_idx]);
          if (dist2 <= radius2) {
            int nid = num_neighbors;
            if (sort_by_dist) {
              // insertion sort
              while ((nid > 0) && (dist2 < dists_ptr[nid-1])) {
                //printf("%d: decreasing nid\n", threadid);
                // move element (nid-1) to place (nid)
                if (nid < max_num_neighbor) {
                  dists_ptr[nid] = dists_ptr[nid-1];
                  edges_ptr[nid*2] = edges_ptr[nid*2-2];
                  edges_ptr[nid*2+1] = edges_ptr[nid*2-1];
                }
                nid--;
              }
            }
            if (nid < max_num_neighbor) {
              edges_ptr[nid*2] = reverse_indices[hash_idx];
              edges_ptr[nid*2+1] = threadid;
              dists_ptr[nid] = dist2;
              num_neighbors++;
            }
            if (num_neighbors > max_num_neighbor) {
              num_neighbors = max_num_neighbor;
            }
            //if (threadid <= 10) {
            //  printf("%d: inserted %" PRId64 ", dist=%f, nid=%d, dists=(%f, %f), num_nbr=%d\n", threadid, reverse_indices[hash_idx], sqrt(dist2), nid, sqrt(dists_ptr[0]), sqrt(dists_ptr[1]), num_neighbors);
            //}
          }
        }
        hash_idx = (hash_idx + 1) % ht_size;
      }
      temp = c;
      for (int i = 0; i < num_dim; i++) {
        query_key_ptr[i] -= temp % (qmax[i] - qmin[i] + 1) + qmin[i];
        temp /= (qmax[i] - qmin[i] + 1);
      }
    }
  }
  __syncthreads();
}

void hash_insert_gpu(at::Tensor keys, at::Tensor values,
                     at::Tensor reverse_indices, at::Tensor dims,
                     at::Tensor insert_keys, at::Tensor insert_values
                     ) {
  CHECK_INPUT(keys);
  CHECK_INPUT(values);
  CHECK_INPUT(reverse_indices);
  CHECK_INPUT(dims);
  CHECK_INPUT(insert_keys);
  CHECK_INPUT(insert_values);

  Key* key_data = keys.data<Key>();
  Key* reverse_indices_data = reverse_indices.data<Key>();
  int num_dim = insert_values.size(1);
  const Key* insert_key_data = insert_keys.data<Key>();
  Float* value_data = values.data<Float>();
  const Float* insert_value_data = insert_values.data<Float>();
  index_t ht_size = keys.size(0);
  uint32 num_inserts = insert_keys.size(0);
  const Key* dims_data = dims.data<Key>();
  int mingridsize, threadblocksize;
  hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize,
     hash_insert_gpu_kernel, 0, 0);

  uint32 gridsize = (num_inserts + threadblocksize - 1) / threadblocksize;
  hash_insert_gpu_kernel<<<gridsize, threadblocksize>>>(
    key_data, value_data, reverse_indices_data, 
    ht_size, dims_data, num_dim,
    insert_key_data, insert_value_data,
    num_inserts
  );
}

void correspondence(at::Tensor keys, at::Tensor values, at::Tensor reverse_indices,
                    at::Tensor dims, at::Tensor query_keys, at::Tensor query_values,
                    at::Tensor qmin, at::Tensor qmax,
                    at::Tensor corres_indices) {
  CHECK_INPUT(keys);
  CHECK_INPUT(values);
  CHECK_INPUT(dims);
  CHECK_INPUT(query_keys);
  CHECK_INPUT(query_values);
  CHECK_INPUT(qmin);
  CHECK_INPUT(qmax);
  CHECK_INPUT(corres_indices);

  Key* key_data = keys.data<Key>();
  Key* reverse_index_data = reverse_indices.data<Key>();
  int num_dim = query_values.size(1);
  Key* query_key_data = query_keys.data<Key>();
  Float* value_data = values.data<Float>();
  const Float* query_value_data = query_values.data<Float>();
  index_t ht_size = keys.size(0);
  uint32 num_queries = query_keys.size(0);
  const int* qmin_data = qmin.data<int>();
  const int* qmax_data = qmax.data<int>();
  const Key* dims_data = dims.data<Key>();

  Key* corres_index_data = corres_indices.data<Key>();

  int mingridsize, threadblocksize;
  hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize,
     correspondence_kernel, 0, 0);

  uint32 gridsize = (num_queries + threadblocksize - 1) / threadblocksize;
  correspondence_kernel<<<gridsize, threadblocksize>>>(
    key_data, value_data, reverse_index_data, 
    ht_size, dims_data, num_dim,
    query_key_data, query_value_data,
    num_queries,
    qmin_data, qmax_data,
    corres_index_data
  );
  
}

torch::Tensor radius_graph_gpu(at::Tensor keys, at::Tensor values, at::Tensor reverse_indices,
                               at::Tensor dims, at::Tensor query_keys, at::Tensor query_values,
                               at::Tensor qmin, at::Tensor qmax,
                               at::Tensor radius, int max_num_neighbors, bool sort_by_dist
                               ) {
  CHECK_INPUT(keys);
  CHECK_INPUT(values);
  CHECK_INPUT(dims);
  CHECK_INPUT(query_keys);
  CHECK_INPUT(query_values);
  CHECK_INPUT(qmin);
  CHECK_INPUT(qmax);
  CHECK_INPUT(radius);

  Key* key_data = keys.data<Key>();
  Key* reverse_index_data = reverse_indices.data<Key>();
  int num_dim = query_values.size(1);
  Key* query_key_data = query_keys.data<Key>();
  Float* value_data = values.data<Float>();
  const Float* query_value_data = query_values.data<Float>();
  index_t ht_size = keys.size(0);
  uint32 num_queries = query_keys.size(0);
  const int* qmin_data = qmin.data<int>();
  const int* qmax_data = qmax.data<int>();
  const Key* dims_data = dims.data<Key>();
  const Float* radius_data = radius.data<Float>();


  int mingridsize, threadblocksize;
  hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize,
     count_radius_graph_degree_kernel, 0, 0);

  int gridsize = (num_queries + threadblocksize - 1) / threadblocksize;
  
  torch::Tensor degree = qmin.new_empty(num_queries);
  int* degree_data = degree.data<int>();

  count_radius_graph_degree_kernel<<<gridsize, threadblocksize>>>(
    key_data, value_data, reverse_index_data,
    ht_size, dims_data, num_dim,
    query_key_data, query_value_data,
    num_queries,
    qmin_data, qmax_data,
    degree_data,
    max_num_neighbors, radius_data
  );

  torch::Tensor offset = cumsum(degree, 0, torch::kInt32);
  offset = offset - degree;
  int* offset_data = offset.data<int>();
  int max_degree = degree.max().item<int>();
  int num_edges = degree.sum().item<int>();

  torch::Tensor edges = keys.new_zeros({num_edges, 2});
  Key* edge_data = edges.data<Key>();
  torch::Tensor dists = values.new_zeros(num_edges);
  Float* dist_data = dists.data<Float>();
  
  radius_graph_kernel<<<gridsize, threadblocksize>>>(
    key_data, value_data, reverse_index_data,
    ht_size, dims_data, num_dim,
    query_key_data, query_value_data,
    num_queries,
    qmin_data, qmax_data,
    degree_data,
    offset_data,
    edge_data,
    radius_data,
    dist_data,
    max_degree,
    sort_by_dist
  );

  return edges;
}

void points_in_radius_gpu(at::Tensor keys, at::Tensor values, at::Tensor reverse_indices,
                          at::Tensor dims, at::Tensor query_keys, at::Tensor query_values,
                          at::Tensor qmin, at::Tensor qmax,
                          Float radius, at::Tensor visited) {
  CHECK_INPUT(keys);
  CHECK_INPUT(values);
  CHECK_INPUT(dims);
  CHECK_INPUT(query_keys);
  CHECK_INPUT(query_values);
  CHECK_INPUT(qmin);
  CHECK_INPUT(qmax);
  CHECK_INPUT(visited);

  Key* key_data = keys.data<Key>();
  Key* reverse_index_data = reverse_indices.data<Key>();
  int num_dim = query_values.size(1);
  Key* query_key_data = query_keys.data<Key>();
  Float* value_data = values.data<Float>();
  const Float* query_value_data = query_values.data<Float>();
  index_t ht_size = keys.size(0);
  uint32 num_queries = query_keys.size(0);
  const int* qmin_data = qmin.data<int>();
  const int* qmax_data = qmax.data<int>();
  const Key* dims_data = dims.data<Key>();

  Key* visited_data = visited.data<Key>();

  int mingridsize, threadblocksize;
  hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize,
     correspondence_kernel, 0, 0);

  uint32 gridsize = (num_queries + threadblocksize - 1) / threadblocksize;
  points_in_radius_kernel<<<gridsize, threadblocksize>>>(
    key_data, value_data, reverse_index_data,
    ht_size, dims_data, num_dim,
    query_key_data, query_value_data,
    num_queries,
    qmin_data, qmax_data,
    radius,
    visited_data
  );
  
}
